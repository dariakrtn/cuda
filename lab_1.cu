#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>

#define	N	(1024*1024)

__global__ void kernel( float* y )
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    float x = 2.0f * 3.1415926f * (float)idx / (float)N;
    y[idx] = logf(x);
}

int main(int argc, char* argv[])
{
    //CPU

    int start2, time2;
    float* data2 = new float[N];

    start2 = clock();

    for (int idx2 = 0; idx2 < N; idx2++)
    {
        float x2 = 2.0f * 3.1415926f * (float)idx2 / (float)N;
        data2[idx2] = logf(x2);
        
    }
    
    time2 = clock() - start2;
    double time_CPU = time2;

    printf("\nCPU Time: %f milliseconds\n", time_CPU);

    //GPU

    float* a = new float[N];
    float* dev = NULL;

    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMalloc((void**)&dev, N * sizeof(float));

    kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);

    hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("GPU Time: %.2f milliseconds\n", gpuTime);

    hipFree(dev);


    return 0;

}
